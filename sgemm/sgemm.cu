#include <hip/hip_runtime.h>
#include <iostream>

template<int X>
__host__ __device__ __inline__ int constexpr getBits() {
  switch (X) {
    #define CASE_X(bits) { case (1<<(bits)): return bits; break; }
    CASE_X(0); CASE_X(1); CASE_X(2); CASE_X(3); CASE_X(4); CASE_X(5); CASE_X(6); CASE_X(7); CASE_X(8); CASE_X(9);
    CASE_X(10); CASE_X(11); CASE_X(12); CASE_X(13); CASE_X(14); CASE_X(15); CASE_X(16); CASE_X(17); CASE_X(18); CASE_X(19);
  }
}

#define OFFSET(row, col, stride) (((row) << (getBits<(stride)>())) + (col))

////////////////////////////////////////////////////////////////////////
// 1. naive
// A: row_major [M, K]
// B: col_major [N, K]
// C: row_major [M, N]
template<int M, int N, int K>
__global__ void sgemm_naive_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId >= M * N) return;

  int constexpr nbits = getBits<N>();
  int constexpr kbits = getBits<K>();
  int m = (threadId >> nbits);
  int n = threadId & (N - 1);
  float vsum = 0.f;
  #pragma unroll
  for (int k = 0; k < K; ++k) {
    // vsum += A[OFFSET(m, k, K)] * B[OFFSET(n, k, N)];
    vsum += A[OFFSET(m, k, K)] * B[OFFSET(k, n, K)];
  }

  C[OFFSET(m, n, N)] = vsum;
}

template<int M, int N, int K>
void _launch_sgemm_naive_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, hipStream_t stream) {
  static_assert((M&31) == 0 && (N&31) == 0 && (K&31) == 0, "M, N, K should be divisible by 32");
  int constexpr blockSz = 128;
  int constexpr gridSz = (M*N + blockSz - 1) / blockSz;
  int constexpr nbits = getBits<N>();
  int constexpr kbits = getBits<K>();
  // std::cout << gridSz << " " << blockSz << " " << nbits << " " << kbits << std::endl;
  sgemm_naive_kernel<M, N, K><<<gridSz, blockSz, 0, stream>>>(A, B, C);
}

////////////////////////////////////////////////////////////////////////
// 2. naive vector
// A: row_major [M, K]
// B: col_major [N, K]
// C: row_major [M, N]
template<int M, int N, int K, typename vecT>
__global__ void sgemm_naive_vec_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId >= M * N) return;

  int constexpr nbits = getBits<N>();
  int constexpr kbits = getBits<K>();
  int constexpr vecSz = sizeof(vecT) / sizeof(float);
  int m = (threadId >> nbits);
  int n = threadId & (N - 1);
  float vsum = 0.f;
  vecT a_val;
  vecT b_val;

  #pragma unroll
  for (int ki = 0; ki < K/vecSz; ++ki) {
    a_val = ((vecT*)A)[(m << getBits<K/vecSz>()) + ki];
    b_val = ((vecT*)B)[(n << getBits<K/vecSz>()) + ki];
    #pragma unroll
    for (int kj = 0; kj < vecSz; ++kj) {
      float a = ((float*)(&a_val))[kj];
      float b = ((float*)(&b_val))[kj];
      vsum += a * b;
    }
  }

  C[(m << nbits) + n] = vsum;
}

template<int M, int N, int K>
void _launch_sgemm_naive_vec_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, hipStream_t stream) {
  static_assert((M&31) == 0 && (N&31) == 0 && (K&31) == 0, "M, N, K should be divisible by 32");
  int constexpr blockSz = 128;
  int constexpr gridSz = (M*N + blockSz - 1) / blockSz;
  sgemm_naive_vec_kernel<M, N, K, double4><<<gridSz, blockSz, 0, stream>>>(A, B, C);
}

////////////////////////////////////////////////////////////////////////
// 3. 2d index
// A: row_major [M, K]
// B: col_major [N, K]
// C: row_major [M, N]
template<int M, int N, int K>
__global__ void sgemm_2dindex_kernel(float* __restrict__ a, float* __restrict__ b, float* __restrict__ c) {
  dim3 constexpr blockSz(32, 32);
  dim3 constexpr gridSz((N + blockSz.x - 1) / blockSz.x, (M + blockSz.y - 1) / blockSz.y);  (void)gridSz;

  int n = blockIdx.x * blockSz.x + threadIdx.x;
  int m = blockIdx.y * blockSz.y + threadIdx.y;
  if (m < M && n < N) {
    float psum = 0.0;
    #pragma unroll
    for (int k = 0; k < K; k++) {
        // psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        psum += a[OFFSET(k, m, M)] * b[OFFSET(k, n, N)];
    }
    c[OFFSET(m, n, N)] = psum;
  }
}

template<int M, int N, int K>
void _launch_sgemm_2dindex_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, hipStream_t stream) {
  static_assert((M&31) == 0 && (N&31) == 0 && (K&31) == 0, "M, N, K should be divisible by 32");
  dim3 constexpr blockSz(32, 32);
  dim3 constexpr gridSz((N + blockSz.x - 1) / blockSz.x, (M + blockSz.y - 1) / blockSz.y);
  sgemm_2dindex_kernel<M, N, K><<<gridSz, blockSz, 0, stream>>>(A, B, C);
}

////////////////////////////////////////////////////////////////////////
// 4. 2d index vector
// A: row_major [M, K]
// B: col_major [N, K]
// C: row_major [M, N]
template<int M, int N, int K, typename vecT>
__global__ void sgemm_2dindex_vec_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C) {
  dim3 constexpr blockSz = {32, 32, 1};
  dim3 constexpr gridSz = {(N + blockSz.x - 1) / blockSz.x, (M + blockSz.y - 1) / blockSz.y, 1};  (void)gridSz;

  int n = blockIdx.x * blockSz.x + threadIdx.x;
  int m = blockIdx.y * blockSz.y + threadIdx.y;
  // if (blockIdx.x == 0 && blockIdx.y == 0) printf("%d %d\n", m, n);
  // printf("%d %d %d %d %d\n", m, n, m >= M, N >= N, m >= M || N >= N);
  if (m >= M || n >= N) return;

  int constexpr nbits = getBits<N>();
  int constexpr kbits = getBits<K>();
  int constexpr vecSz = sizeof(vecT) / sizeof(float);

  float vsum = 0.f;
  vecT a_val;
  vecT b_val;

  #pragma unroll
  for (int ki = 0; ki < K/vecSz; ++ki) {
    a_val = ((vecT*)A)[(m << getBits<K/vecSz>()) + ki];
    b_val = ((vecT*)B)[(n << getBits<K/vecSz>()) + ki];
    #pragma unroll
    for (int kj = 0; kj < vecSz; ++kj) {
      float a = ((float*)(&a_val))[kj];
      float b = ((float*)(&b_val))[kj];
      vsum += a * b;
    }
  }

  C[(m << nbits) + n] = vsum;
}

template<int M, int N, int K>
void _launch_sgemm_2dindex_vec_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, hipStream_t stream) {
  static_assert((M&31) == 0 && (N&31) == 0 && (K&31) == 0, "M, N, K should be divisible by 32");
  dim3 constexpr blockSz = {32, 32, 1};
  dim3 constexpr gridSz = {(N + blockSz.x - 1) / blockSz.x, (M + blockSz.y - 1) / blockSz.y, 1};
  sgemm_2dindex_vec_kernel<M, N, K, double2><<<gridSz, blockSz, 0, stream>>>(A, B, C);
}


////////////////////////////////////////////////////////////////////////
void sgemm_cuda(float* __restrict__ dInput, float* __restrict__ dWeight, float* __restrict__ dInputTrans, float* __restrict__ dWeightTrans, float* __restrict__ dOutput, int M, int N, int K, hipStream_t stream) {
  #define IF_STAT if (false)
  #define ELIF_STAT(m, n, k) else if ((m) == M && (n) == N && (k) == K) _launch_sgemm_naive_kernel<(m), (n), (k)>(dInput, dWeightTrans, dOutput, stream)
  // #define ELIF_STAT(m, n, k) else if ((m) == M && (n) == N && (k) == K) _launch_sgemm_naive_vec_kernel<(m), (n), (k)>(dInput, dWeight, dOutput, stream)
  // #define ELIF_STAT(m, n, k) else if ((m) == M && (n) == N && (k) == K) _launch_sgemm_2dindex_kernel<(m), (n), (k)>(dInput, dWeightTrans, dOutput, stream)
  // #define ELIF_STAT(m, n, k) else if ((m) == M && (n) == N && (k) == K) _launch_sgemm_2dindex_vec_kernel<(m), (n), (k)>(dInput, dWeight, dOutput, stream)
  #define ELSE_STAT else { std::cout << "NOT_IMPLEMENTED" << std::endl; __builtin_trap(); }

  IF_STAT;
  ELIF_STAT(32, 32, 32);
  ELIF_STAT(128, 128, 128);
  ELIF_STAT(1024, 1024, 1024);
  ELSE_STAT;

  #undef IF_STAT
  #undef ELIF_STAT
  #undef ELSE_STAT

  // cudaStreamSynchronize(stream);
}