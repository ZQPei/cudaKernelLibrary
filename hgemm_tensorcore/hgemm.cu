#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

template<int X>
__host__ __device__ __inline__ int constexpr getBits() {
  switch (X) {
    #define CASE_X(bits) { case (1<<(bits)): return bits; break; }
    CASE_X(0); CASE_X(1); CASE_X(2); CASE_X(3); CASE_X(4); CASE_X(5); CASE_X(6); CASE_X(7); CASE_X(8); CASE_X(9);
    CASE_X(10); CASE_X(11); CASE_X(12); CASE_X(13); CASE_X(14); CASE_X(15); CASE_X(16); CASE_X(17); CASE_X(18); CASE_X(19);
    #undef CASE_X
    default: return 0;
  }
}

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z
#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z
// #define OFFSET(row, col, stride) (((row) << (getBits<(stride)>())) + (col))
#define OFFSET(row, col, stride) (((row) * stride) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

////////////////////////////////////////////////////////////////////////
// 1. hgemm tensorcore
// A: row_major [M, K]
// B: row_major [K, N]
// C: row_major [M, N]
template<int M, int N, int K,int BM, int BN, int BK>
__global__ void hgemm_tensorcore_kernel(half* __restrict__ A, half* __restrict__ B, half* __restrict__ C) {

}

template<int M, int N, int K>
void _launch_hgemm_tensorcore_kernel(half* __restrict__ A, half* __restrict__ B, half* __restrict__ C, hipStream_t stream) {
  int constexpr BM = 128;
  int constexpr BK = 32;
  int constexpr BN = 256;
  static_assert((M&(BM-1))==0 && (N&(BN-1))==0 && (K&(BK-1))==0, "M, N, K shape mismatch");

  int constexpr blockSz = 256;
  dim3 const gridSz = {N/BN, M/BM};
  hgemm_tensorcore_kernel<M,N,K,BM,BN,BK><<<gridSz, blockSz, 0, stream>>>(A, B, C);
}


////////////////////////////////////////////////////////////////////////
// benchmark: cublas
// A: row_major [M, K]
// B: row_major [K, N]
// C: row_major [M, N]
// cublas
template<int M, int N, int K>
void _launch_hgemm_cudnn_kernel(half* __restrict__ a, half* __restrict__ b, half* __restrict__ c, hipStream_t stream) {
  static hipblasHandle_t cublas_handle = nullptr;
  if (cublas_handle == nullptr) {
    hipblasCreate(&cublas_handle);
    hipblasSetStream(cublas_handle, stream);
  }
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  half alpha = 1.0;
  half beta = 0;
  hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
        &alpha, b, HIP_R_16F, N, a, HIP_R_16F, K, &beta, c, HIP_R_16F, N,
        HIPBLAS_COMPUTE_16F, algo);
}


////////////////////////////////////////////////////////////////////////
void hgemm_cuda(half* __restrict__ A, half* __restrict__ B, half* __restrict__ AT, half* __restrict__ BT, half* __restrict__ C, int M, int N, int K, hipStream_t stream) {
  #define IF_STAT if (false)
  // #define ELIF_STAT(m, n, k) else if ((m) == M && (n) == N && (k) == K) _launch_hgemm_tensorcore_kernel<(m), (n), (k)>(A, B, C, stream)
  #define ELIF_STAT(m, n, k) else if ((m) == M && (n) == N && (k) == K) _launch_hgemm_cudnn_kernel<(m), (n), (k)>(A, B, C, stream)
  #define ELSE_STAT else { std::cout << "NOT_IMPLEMENTED" << std::endl; __builtin_trap(); }

  IF_STAT;
  ELIF_STAT(32, 32, 32);
  ELIF_STAT(128, 128, 128);
  ELIF_STAT(1024, 1024, 1024);
  ELIF_STAT(640000, 128, 32);
  ELIF_STAT(640000, 32, 16);
  ELSE_STAT;

  #undef IF_STAT
  #undef ELIF_STAT
  #undef ELSE_STAT

  // cudaStreamSynchronize(stream);
}
